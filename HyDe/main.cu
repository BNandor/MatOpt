#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "core/common/Constants.cuh"
#include "core/optimizer/LBFGS.cuh"
#include "core/optimizer/GradientDescent.cuh"
#include "core/optimizer/DifferentialEvolution.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <fstream>

void generateInitialPopulation(double *x, unsigned xSize) {
    std::uniform_real_distribution<double> unif(-10000, 10000);
    std::default_random_engine re(time(NULL));
    for (int i = 0; i < xSize; i++) {
        x[i] = unif(re);
    }
}
void readPopulation(double *x, unsigned xSize, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> x[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << xSize
                  << std::endl;
        assert(cData == xSize);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)

void readSNLPProblem(double *data, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> data[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << RESIDUAL_CONSTANTS_COUNT_1 * RESIDUAL_CONSTANTS_DIM_1
                  << std::endl;
        assert(cData == RESIDUAL_CONSTANTS_COUNT_1 * RESIDUAL_CONSTANTS_DIM_1);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

void readSNLPAnchors(double *data, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> data[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << RESIDUAL_CONSTANTS_COUNT_2 * RESIDUAL_CONSTANTS_DIM_2
                  << std::endl;
        assert(cData == RESIDUAL_CONSTANTS_COUNT_2 * RESIDUAL_CONSTANTS_DIM_2);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

#endif
void persistBestSNLPModel(double *x, int modelSize, std::string filename) {
    std::ofstream output;
    output.open(filename.c_str());
    if (output.is_open()) {
        for (int i=0;i<modelSize;i++){
            output<<std::setprecision(17)<<x[i]<<std::endl;
        }
        output.close();
    } else {
        std::cout << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

void testH_DE_SNLP() {

    hiprandState *dev_curandState;
    hipEvent_t start, stop, startCopy, stopCopy;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCopy);
    hipEventCreate(&stopCopy);


    const unsigned xSize = X_DIM * POPULATION_SIZE;

#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)
    const unsigned dataSize = RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1 +
                              RESIDUAL_CONSTANTS_DIM_2 * RESIDUAL_CONSTANTS_COUNT_2;
#else
    const unsigned dataSize = RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1;
#endif
    OPTIMIZER::GlobalData *dev_globalContext;
    hipMalloc(&dev_globalContext, sizeof(OPTIMIZER::GlobalData)*POPULATION_SIZE);
    printf("Allocating %lu global memory\n",sizeof(OPTIMIZER::GlobalData)*POPULATION_SIZE);

    double *dev_x;
    double *dev_xDE;
    double *dev_x1;
    double *dev_x2;
    double *dev_data;
    double *dev_F;
    double *dev_FDE;
    double *dev_F1;
    double *dev_F2;

    // ALLOCATE DEVICE MEMORY
    hipMalloc((void **) &dev_x, xSize * sizeof(double));
    hipMalloc((void **) &dev_xDE, xSize * sizeof(double));
    hipMalloc((void **) &dev_data, dataSize * sizeof(double));
    hipMalloc((void **) &dev_F, POPULATION_SIZE * sizeof(double));
    hipMalloc((void **) &dev_FDE, POPULATION_SIZE * sizeof(double));
    hipMalloc(&dev_curandState, THREADS_PER_GRID * sizeof(hiprandState));

    // GENERATE PROBLEM
    double x[xSize] = {};
    double solution[xSize] = {};
    double finalFs[POPULATION_SIZE] = {};
    double data[dataSize] = {};


#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)
    readSNLPProblem(data, PROBLEM_PATH);

    readSNLPAnchors(data + RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1,
                    PROBLEM_ANCHOR_PATH);
//    generateInitialPopulation(x, xSize);
    readPopulation(x, xSize,PROBLEM_INPUT_POPULATION_PATH);
#endif
    // COPY TO DEVICE
    hipEventRecord(startCopy);
    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_data, &data, dataSize * sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(stopCopy);
    hipEventRecord(start);

    // EXECUTE KERNEL
    // initialize hiprand
    setupCurand<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_curandState);
    dev_x1 = dev_x;
    dev_x2 = dev_xDE;
    dev_F1 = dev_F;
    dev_F2 = dev_FDE;

#if  defined(OPTIMIZER_MIN_INIT_DE) || defined(OPTIMIZER_MIN_DE)
    OPTIMIZER::optimize<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_data,dev_F1, dev_globalContext);
#endif

#ifdef OPTIMIZER_SIMPLE_DE
    OPTIMIZER::evaluateF<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_data, dev_F1, dev_globalContext);
#endif

    for (unsigned i = 0; i < DE_ITERATION_COUNT; i++) {
        differentialEvolutionStep<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_x2, dev_curandState);
        //dev_x2 is the differential model
#if  defined(OPTIMIZER_MIN_INIT_DE) || defined(OPTIMIZER_SIMPLE_DE)
        OPTIMIZER::evaluateF<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x2, dev_data, dev_F2, dev_globalContext);
#elif defined(OPTIMIZER_MIN_DE)
        OPTIMIZER::optimize<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x2, dev_data, dev_F2, dev_globalContext);
#else
        std::cerr<<"Incorrect optimizer configuration"<<std::endl;
        exit(1);
#endif
        //evaluated differential model into F2
        selectBestModels<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_x2, dev_F1, dev_F2, i);
        //select the best models from current and differential models
        std::swap(dev_x1, dev_x2);
        std::swap(dev_F1, dev_F2);
        // dev_x1 contains the next models, dev_F1 contains the associated costs
    }
#if defined(OPTIMIZER_SIMPLE_DE) || defined(OPTIMIZER_MIN_INIT_DE) || defined(OPTIMIZER_MIN_DE)
        printf("\nthreads:%d\n", THREADS_PER_BLOCK);
        printf("\niterations:%d\n", DE_ITERATION_COUNT);
        printf("\nfevaluations: %d\n", DE_ITERATION_COUNT);
#endif

    printBestF<<<1,1>>>(dev_F1,POPULATION_SIZE);

    hipMemcpy(&finalFs, dev_F1, POPULATION_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    int min=0;
    for(int ff=1;ff<POPULATION_SIZE;ff++){
        if(finalFs[min]>finalFs[ff]){
            min=ff;
        }
    }
    hipMemcpy(&solution, dev_x1, xSize * sizeof(double), hipMemcpyDeviceToHost);
    printf("\nsolf: %f and solution: ",finalFs[min]);
    for(int ff=X_DIM*min;ff<X_DIM*(min+1)-1;ff++) {
        printf("%f,",solution[ff]);
    }
    printf("%f\n",solution[X_DIM*(min+1)-1]);
    persistBestSNLPModel(&solution[X_DIM*min],X_DIM, std::string("finalModel")+std::string(OPTIMIZER::name)+std::string(".csv"));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float memcpyMilli = 0;
    hipEventElapsedTime(&memcpyMilli, startCopy, stopCopy);
    float kernelMilli = 0;
    hipEventElapsedTime(&kernelMilli, start, stop);
//    printf("Memcpy,kernel elapsed time (ms): %f,%f\n", memcpyMilli, kernelMilli);
    printf("\ntime ms : %f\n", kernelMilli);


    hipFree(dev_x);
    hipFree(dev_xDE);
    hipFree(dev_data);
    hipFree(dev_F);
    hipFree(dev_FDE);

#ifdef GLOBAL_SHARED_MEM
    hipFree(dev_globalContext);
#endif
}

int main() {
    testH_DE_SNLP();
    return 0;
}